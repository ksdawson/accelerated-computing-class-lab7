#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

template <typename Op>
void print_array(
    size_t n,
    typename Op::Data const *x // allowed to be either a CPU or GPU pointer
);

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

template <typename Op>
void scan_cpu(size_t n, typename Op::Data const *x, typename Op::Data *out) {
    using Data = typename Op::Data;
    Data accumulator = Op::identity();
    for (size_t i = 0; i < n; i++) {
        accumulator = Op::combine(accumulator, x[i]);
        out[i] = accumulator;
    }
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

/// Helpers to deal with Op::Data type

// Generic, aligned struct for vectorized memory access
template <typename T, int N>
struct alignas(sizeof(T) * N) Vectorized {
    T elements[N];
};

// Needed because compiler doesn't know how to shuffle DebugRange
template <typename T>
__device__ T shfl_up_any(T val, unsigned int delta) {
    T result;
    if constexpr (sizeof(T) == 4) {
        // Single 32-bit value
        uint32_t v = *reinterpret_cast<uint32_t*>(&val);
        v = __shfl_up_sync(0xffffffff, v, delta);
        *reinterpret_cast<uint32_t*>(&result) = v;
    } else {
        // Two 32-bit values (e.g. DebugRange)
        const uint32_t* src = reinterpret_cast<const uint32_t*>(&val);
        uint32_t* dst = reinterpret_cast<uint32_t*>(&result);
        dst[0] = __shfl_up_sync(0xffffffff, src[0], delta);
        dst[1] = __shfl_up_sync(0xffffffff, src[1], delta);
    }
    return result;
}

namespace scan_gpu {

// Helpers
// template <typename Op, uint32_t VEC_SIZE>
// __device__ void thread_local_scan(void *val, typename Op::Data seed) {
//     // Vectorize pointer
//     using VecData = Vectorized<Data, VEC_SIZE>;
//     VecData valVec = reinterpret_cast<VecData*>(val);

//     // Compute a local scan for a vector of Data for each thread
//     valVec.elements[0] = Op::combine(seed, valVec.elements[0]);
//     #pragma unroll
//     for (uint32_t i = 1; i < VEC_SIZE; ++i) {
//         valVec.elements[i] = Op::combine(valVec.elements[i-1], valVec.elements[i]);
//     }
// }

template <typename Op>
__device__ typename Op::Data warp_local_scan(typename Op::Data val) {
    using Data = typename Op::Data;

    // Computes parallel prefix on 32 elements using Hillis Steele Scan w/ warp shuffle
    const uint32_t thread_idx = threadIdx.x % 32;
    uint32_t idx = 1;
    for (uint32_t step = 0; step < 5; ++step) { // log2(32) = 5
        // Load prefix from register
        Data tmp = shfl_up_any(val, idx);
        tmp = (thread_idx >= idx) ? tmp : Op::identity(); // Mask out

        // Update prefix in register
        val = Op::combine(tmp, val);

        // Multiply idx by 2
        idx <<= 1;
    }

    return val;
}

// __device__ void sm_scan() {
//     // New approach for intra-block
//     // (1) Load block from GMEM to SMEM
//     // (2) Each thread gets a contiguous block from SMEM
//     // (3) Warp shuffle endpoints
// }

// template <typename Op, uint32_t VEC_SIZE, bool FIX>
// __device__ void sm_scan(size_t n, typename Op::Data *in, typename Op::Data *out) {
//     // Data types
//     using Data = typename Op::Data;
//     using VecData = Vectorized<Data, VEC_SIZE>;

//     // Thread block info
//     const uint32_t num_warps = blockDim.x / 32;
//     const uint32_t warp_idx = threadIdx.x / 32;
//     const uint32_t thread_idx = threadIdx.x % 32;

//     //
//     constexpr uint32_t warp_block_size = 32 * VEC_SIZE;
//     constexpr uint32_t out_n = n / warp_block_size;

//     // First pass local scan to get the endpoint for each warp w/ no seed
//     for (uint32_t idx = warp_idx; idx < out_n; idx += num_warps) {
//         // Move buffer
//         Data *win = in + idx * warp_block_size;

//         // Vector load from memory
//         VecData valVec = reinterpret_cast<VecData*>(win)[thread_idx];

//         // Compute a local scan for a vector of Data for each thread
//         thread_local_scan((void *)&valVec, Op::identity());

//         // Compute a hierarchical scan on the endpoints from each thread scan
//         Data end = valVec.elements[VEC_SIZE - 1];
//         Data end_fixed = warp_local_scan<Op>(end);

//         // Write only the last to memory
//         if (thead_idx == 31) {
//             out[idx] = end_fixed;
//         }
//     }

//     // Hierarchical scan on the endpoints for each warp
//     Data *new_out = out + out_n;
//     sm_scan(n / warp_block_size, out, new_out);

//     // Second pass local scan fix to get all points w/ a seed
//     if constexpr (FIX) {
        
//     }
// }

// // Kernel stages: local, hierarchical, local fix
// template <typename Op>
// __global__ void local_scan(size_t n, typename Op::Data *x, void *workspace) {
//     using Data = typename Op::Data;

//     // Each SM gets a piece of x
//     size_t sm_n = n / gridDim.x;
//     // Handle tail by giving the rest to the last SM
//     sm_n += (blockIdx.x == gridDim.x - 1) ? n % gridDim.x : 0;

//     // Make tmp x
//     Data *tmp_x = reinterpret_cast<Data*>(workspace); // TODO: Switch to SMEM

//     // Move buffers
//     x += blockIdx.x * sm_n;
//     tmp_x += blockIdx.x * (sm_n / 128 + sm_n/256);

//     // Thread block info
//     const uint32_t num_warps = blockDim.x / 32;
//     const uint32_t warp_idx = threadIdx.x / 32;
//     const uint32_t thread_idx = threadIdx.x % 32;

//     // Compute first level
//     for (uint32_t idx = warp_idx; idx < sm_n / 128; idx += num_warps) {
//         // Move buffer
//         Data *wx = x + idx * 128;
//         // Local scan on the warp chunk
//         Data end = warp_local_scan(wx);
//         // If last thread write end back to tmp_x
//         if (thread_idx == 31) {
//             tmp_x[idx] = end;
//         }
//     }

//     // Setup buffers
//     std::swap(x, tmp_x);


//     // Iterate over hierarchy
//     while (sm_n > 0) {
//         // Iterate over blocks of 128 at this level
//         for (uint32_t idx = warp_idx; idx < sm_n / 128; idx += num_warps) {
//             // Move buffer
//             Data *wx = x + idx * 128;
//             // Local scan on the warp chunk
//             Data end = warp_local_scan(wx);
//             // If last thread write end back to tmp_x
//             if (thread_idx == 31) {
//                 tmp_x[idx] = end;
//             }
//         }

//         // Setup next level
//         sm_n /= 128;
//         std::swap(x, tmp_x);

//         // Wait for this level to be done
//         __syncthreads();
//     }
// }

// template <typename Op>
// __global__ void hierarchical_scan(size_t n, typename Op::Data *x, void *workspace) {
//     using Data = typename Op::Data;
// }

// template <typename Op>
// __global__ void local_scan_fix(size_t n, typename Op::Data *x, void *workspace) {
//     using Data = typename Op::Data;
// }

// Sequential Kernels
template <typename Op>
__global__ void scan_gpu_single_thread(size_t n, typename Op::Data const *x, typename Op::Data *out) {
    using Data = typename Op::Data;
    Data accumulator = Op::identity();
    for (size_t i = 0; i < n; i++) {
        accumulator = Op::combine(accumulator, x[i]);
        out[i] = accumulator;
    }
}
template <typename Op, uint32_t VEC_SIZE>
__global__ void scan_gpu_single_warp(size_t n, typename Op::Data const *x, typename Op::Data *out) {
    // Data types
    using Data = typename Op::Data;
    using VecData = Vectorized<Data, VEC_SIZE>;

    // Work for each thread
    const uint32_t thread_idx = threadIdx.x % 32;
    const uint32_t n_per_thread = n / 32;
    const uint32_t start_i = threadIdx.x * n_per_thread;
    const uint32_t end_i = (threadIdx.x + 1) * n_per_thread;
    const uint32_t start_vi = start_i / VEC_SIZE;
    const uint32_t end_vi = end_i / VEC_SIZE;

    // Vectorize
    VecData const *vx = reinterpret_cast<VecData const *>(x);
    VecData *vout = reinterpret_cast<VecData*>(out);

    // Local scan
    Data accumulator = Op::identity();
    for (uint32_t i = start_vi; i < end_vi; ++i) {
        VecData v = vx[i];
        #pragma unroll
        for (uint32_t vi = 0; vi < VEC_SIZE; ++vi) {
            accumulator = Op::combine(accumulator, v.elements[vi]);
        }
    }
    __syncwarp();

    // Hierarchical scan on endpoints
    accumulator = warp_local_scan<Op>(accumulator);

    // Shuffle accumulators
    accumulator = shfl_up_any(accumulator, 1);
    accumulator = (thread_idx >= 1) ? accumulator : Op::identity();

    // Local scan fix
    for (uint32_t i = start_vi; i < end_vi; ++i) {
        VecData v = vx[i];
        #pragma unroll
        for (uint32_t vi = 0; vi < VEC_SIZE; ++vi) {
            accumulator = Op::combine(accumulator, v.elements[vi]);
            v.elements[vi] = accumulator;
        }
        // Output to memory
        vout[i] = v;
    }

    // Handle tail
    if (thread_idx == 31) {
        for (uint32_t i = end_i; i < end_i + n % 32; ++i) {
            accumulator = Op::combine(accumulator, x[i]);
            out[i] = accumulator;
        }
    }
}

// Returns desired size of scratch buffer in bytes.
template <typename Op> size_t get_workspace_size(size_t n) {
    using Data = typename Op::Data;
    /* TODO: your CPU code here... */
    return 0;
}

// 'launch_scan'
//
// Input:
//
//   'n': Number of elements in the input array 'x'.
//
//   'x': Input array in GPU memory. The 'launch_scan' function is allowed to
//   overwrite the contents of this buffer.
//
//   'workspace': Scratch buffer in GPU memory. The size of the scratch buffer
//   in bytes is determined by 'get_workspace_size<Op>(n)'.
//
// Output:
//
//   Returns a pointer to GPU memory which will contain the results of the scan
//   after all launched kernels have completed. Must be either a pointer to the
//   'x' buffer or to an offset within the 'workspace' buffer.
//
//   The contents of the output array should be "partial reductions" of the
//   input; each element 'i' of the output array should be given by:
//
//     output[i] = Op::combine(x[0], x[1], ..., x[i])
//
//   where 'Op::combine(...)' of more than two arguments is defined in terms of
//   repeatedly combining pairs of arguments. Note that 'Op::combine' is
//   guaranteed to be associative, but not necessarily commutative, so
//
//        Op::combine(a, b, c)              // conceptual notation; not real C++
//     == Op::combine(a, Op::combine(b, c)) // real C++
//     == Op::combine(Op::combine(a, b), c) // real C++
//
//  but we don't necessarily have
//
//    Op::combine(a, b) == Op::combine(b, a) // not true in general!
//
template <typename Op>
typename Op::Data *launch_scan(
    size_t n,
    typename Op::Data *x, // pointer to GPU memory
    void *workspace       // pointer to GPU memory
) {
    using Data = typename Op::Data;

    if (sizeof(Data) == 4) {
        scan_gpu_single_warp<Op, 4><<<1, 32>>>(n, x, x);
        return x;
    } else if (sizeof(Data) == 8) {
        scan_gpu_single_warp<Op, 2><<<1, 32>>>(n, x, x);
        return x;
    } else {
        return nullptr;
    }
}

} // namespace scan_gpu

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

struct DebugRange {
    uint32_t lo;
    uint32_t hi;

    static constexpr uint32_t INVALID = 0xffffffff;

    static __host__ __device__ __forceinline__ DebugRange invalid() {
        return {INVALID, INVALID};
    }

    __host__ __device__ __forceinline__ bool operator==(const DebugRange &other) const {
        return lo == other.lo && hi == other.hi;
    }

    __host__ __device__ __forceinline__ bool operator!=(const DebugRange &other) const {
        return !(*this == other);
    }

    __host__ __device__ bool is_empty() const { return lo == hi; }

    __host__ __device__ bool is_valid() const { return lo != INVALID; }

    std::string to_string() const {
        if (lo == INVALID) {
            return "INVALID";
        } else {
            return std::to_string(lo) + ":" + std::to_string(hi);
        }
    }
};

struct DebugRangeConcatOp {
    using Data = DebugRange;

    static __host__ __device__ __forceinline__ Data identity() { return {0, 0}; }

    static __host__ __device__ __forceinline__ Data combine(Data a, Data b) {
        if (a.is_empty()) {
            return b;
        } else if (b.is_empty()) {
            return a;
        } else if (a.is_valid() && b.is_valid() && a.hi == b.lo) {
            return {a.lo, b.hi};
        } else {
            return Data::invalid();
        }
    }

    static std::string to_string(Data d) { return d.to_string(); }
};

struct SumOp {
    using Data = uint32_t;

    static __host__ __device__ __forceinline__ Data identity() { return 0; }

    static __host__ __device__ __forceinline__ Data combine(Data a, Data b) {
        return a + b;
    }

    static std::string to_string(Data d) { return std::to_string(d); }
};

constexpr size_t max_print_array_output = 1025;
static thread_local size_t total_print_array_output = 0;

template <typename Op> void print_array(size_t n, typename Op::Data const *x) {
    using Data = typename Op::Data;

    // copy 'x' from device to host if necessary
    hipPointerAttribute_t attr;
    CUDA_CHECK(hipPointerGetAttributes(&attr, x));
    auto x_host_buf = std::vector<Data>();
    Data const *x_host_ptr = nullptr;
    if (attr.type == hipMemoryTypeDevice) {
        x_host_buf.resize(n);
        x_host_ptr = x_host_buf.data();
        CUDA_CHECK(
            hipMemcpy(x_host_buf.data(), x, n * sizeof(Data), hipMemcpyDeviceToHost));
    } else {
        x_host_ptr = x;
    }

    if (total_print_array_output >= max_print_array_output) {
        return;
    }

    printf("[\n");
    for (size_t i = 0; i < n; i++) {
        auto s = Op::to_string(x_host_ptr[i]);
        printf("  [%zu] = %s,\n", i, s.c_str());
        total_print_array_output++;
        if (total_print_array_output > max_print_array_output) {
            printf("  ... (output truncated)\n");
            break;
        }
    }
    printf("]\n");

    if (total_print_array_output >= max_print_array_output) {
        printf("(Reached maximum limit on 'print_array' output; skipping further calls "
               "to 'print_array')\n");
    }

    total_print_array_output++;
}

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}

struct Results {
    double time_ms;
    double bandwidth_gb_per_sec;
};

enum class Mode {
    TEST,
    BENCHMARK,
};

template <typename Op>
Results run_config(Mode mode, std::vector<typename Op::Data> const &x) {
    // Allocate buffers
    using Data = typename Op::Data;
    size_t n = x.size();
    size_t workspace_size = scan_gpu::get_workspace_size<Op>(n);
    Data *x_gpu;
    Data *workspace_gpu;
    CUDA_CHECK(hipMalloc(&x_gpu, n * sizeof(Data)));
    CUDA_CHECK(hipMalloc(&workspace_gpu, workspace_size));
    CUDA_CHECK(hipMemcpy(x_gpu, x.data(), n * sizeof(Data), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));

    // Test correctness
    auto expected = std::vector<Data>(n);
    scan_cpu<Op>(n, x.data(), expected.data());
    auto out_gpu = scan_gpu::launch_scan<Op>(n, x_gpu, workspace_gpu);
    if (out_gpu == nullptr) {
        printf("'launch_scan' function not yet implemented (returned nullptr)\n");
        exit(1);
    }
    auto actual = std::vector<Data>(n);
    CUDA_CHECK(
        hipMemcpy(actual.data(), out_gpu, n * sizeof(Data), hipMemcpyDeviceToHost));
    for (size_t i = 0; i < n; ++i) {
        if (actual.at(i) != expected.at(i)) {
            auto actual_str = Op::to_string(actual.at(i));
            auto expected_str = Op::to_string(expected.at(i));
            printf(
                "Mismatch at position %zu: %s != %s\n",
                i,
                actual_str.c_str(),
                expected_str.c_str());
            if (n <= 128) {
                printf("Input:\n");
                print_array<Op>(n, x.data());
                printf("\nExpected:\n");
                print_array<Op>(n, expected.data());
                printf("\nActual:\n");
                print_array<Op>(n, actual.data());
            }
            exit(1);
        }
    }
    if (mode == Mode::TEST) {
        return {0.0, 0.0};
    }

    // Benchmark
    double target_time_ms = 200.0;
    double time_ms = benchmark_ms(
        target_time_ms,
        [&]() {
            CUDA_CHECK(
                hipMemcpy(x_gpu, x.data(), n * sizeof(Data), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
        },
        [&]() { scan_gpu::launch_scan<Op>(n, x_gpu, workspace_gpu); });
    double bytes_processed = n * sizeof(Data) * 2;
    double bandwidth_gb_per_sec = bytes_processed / time_ms / 1e6;

    // Cleanup
    CUDA_CHECK(hipFree(x_gpu));
    CUDA_CHECK(hipFree(workspace_gpu));

    return {time_ms, bandwidth_gb_per_sec};
}

std::vector<DebugRange> gen_debug_ranges(uint32_t n) {
    auto ranges = std::vector<DebugRange>();
    for (uint32_t i = 0; i < n; ++i) {
        ranges.push_back({i, i + 1});
    }
    return ranges;
}

template <typename Rng> std::vector<uint32_t> gen_random_data(Rng &rng, uint32_t n) {
    auto uniform = std::uniform_int_distribution<uint32_t>(0, 100);
    auto data = std::vector<uint32_t>();
    for (uint32_t i = 0; i < n; ++i) {
        data.push_back(uniform(rng));
    }
    return data;
}

template <typename Op, typename GenData>
void run_tests(std::vector<uint32_t> const &sizes, GenData &&gen_data) {
    for (auto size : sizes) {
        auto data = gen_data(size);
        printf("  Testing size %8u\n", size);
        run_config<Op>(Mode::TEST, data);
        printf("  OK\n\n");
    }
}

int main(int argc, char const *const *argv) {
    auto correctness_sizes = std::vector<uint32_t>{
        16,
        10,
        128,
        100,
        1024,
        1000,
        // 1 << 20,
        // 1'000'000,
        // 16 << 20,
        // 64 << 20,
    };

    auto rng = std::mt19937(0xCA7CAFE);

    printf("Correctness:\n\n");
    printf("Testing scan operation: debug range concatenation\n\n");
    run_tests<DebugRangeConcatOp>(correctness_sizes, gen_debug_ranges);
    printf("Testing scan operation: integer sum\n\n");
    run_tests<SumOp>(correctness_sizes, [&](uint32_t n) {
        return gen_random_data(rng, n);
    });

    printf("Performance:\n\n");

    size_t n = 64 << 20;
    auto data = gen_random_data(rng, n);

    printf("Benchmarking scan operation: integer sum, size %zu\n\n", n);

    // Warmup
    run_config<SumOp>(Mode::BENCHMARK, data);
    // Benchmark
    auto results = run_config<SumOp>(Mode::BENCHMARK, data);
    printf("  Time: %.2f ms\n", results.time_ms);
    printf("  Throughput: %.2f GB/s\n", results.bandwidth_gb_per_sec);

    return 0;
}