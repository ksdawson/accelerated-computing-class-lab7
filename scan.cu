#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

template <typename Op>
void print_array(
    size_t n,
    typename Op::Data const *x // allowed to be either a CPU or GPU pointer
);

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

template <typename Op>
void scan_cpu(size_t n, typename Op::Data const *x, typename Op::Data *out) {
    using Data = typename Op::Data;
    Data accumulator = Op::identity();
    for (size_t i = 0; i < n; i++) {
        accumulator = Op::combine(accumulator, x[i]);
        out[i] = accumulator;
    }
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

namespace scan_gpu {

// Generic, aligned struct for vectorized memory access
template <typename T, int N>
struct alignas(sizeof(T) * N) Vectorized {
    T elements[N];
};

// Helpers
template <typename Op>
__device__ typename Op::Data unit_local_scan(typename Op::Data val) {
    using Data = typename Op::Data;
    // Computes parallel prefix on 32 elements using Hillis Steele Scan w/ warp shuffle
    const uint32_t tidx = threadIdx.x % 32;
    uint32_t idx = 1;
    for (uint32_t step = 0; step < 5; ++step) { // log2(32) = 5
        // Load prefix from register
        Data tmp = __shfl_up_sync(0xffffffff, val, idx);
        tmp = (tidx >= idx) ? tmp : Op::identity(); // Mask out
        // Update prefix in register
        val = Op::combine(val, tmp);
        // Multiply idx by 2
        idx <<= 1;
    }
    return val;
}

template <typename Op, uint32_t VEC_SIZE>
__device__ typename Op::Data warp_local_scan(typename Op::Data *x) {
    // Data types
    using Data = typename Op::Data;
    using VecData = Vectorized<Data, VEC_SIZE>;

    // Vector load from memory
    const uint32_t tidx = threadIdx.x % 32;
    VecData valVec = reinterpret_cast<VecData*>(x)[tidx];

    // Compute a local scan for a vector of Data for each thread
    #pragma unroll
    for (uint32_t i = 1; i < VEC_SIZE; ++i) {
        valVec.elements[i] = Op::combine(valVec.elements[i-1], valVec.elements[i]);
    }

    // Compute a hierarchical scan on the endpoints from each thread scan
    Data val = valVec.elements[VEC_SIZE - 1];
    val = unit_local_scan<Op>(val);

    // val for tidx == 31 will be the endpoint for the warp local scan
    return val;
}

// Kernel stages: local, hierarchical, local fix
template <typename Op>
__global__ void local_scan(size_t n, typename Op::Data *x, void *workspace) {
    using Data = typename Op::Data;

    // Each SM gets a piece of x
    size_t sm_n = n / gridDim.x;
    // Handle tail by giving the rest to the last SM
    sm_n += (blockIdx.x == gridDim.x - 1) ? n % gridDim.x : 0;

    // Make tmp x
    Data *tmp_x = reinterpret_cast<Data*>(workspace); // TODO: Switch to SMEM

    // Move buffers
    x += blockIdx.x * sm_n;
    tmp_x += blockIdx.x * (sm_n / 128 + sm_n/256);

    // Thread block info
    const uint32_t num_warps = blockDim.x / 32;
    const uint32_t warp_idx = threadIdx.x / 32;
    const uint32_t thread_idx = threadIdx.x % 32;

    // Compute first level
    for (uint32_t idx = warp_idx; idx < sm_n / 128; idx += num_warps) {
        // Move buffer
        Data *wx = x + idx * 128;
        // Local scan on the warp chunk
        Data end = warp_local_scan(wx);
        // If last thread write end back to tmp_x
        if (thread_idx == 31) {
            tmp_x[idx] = end;
        }
    }

    // Setup buffers
    std::swap(x, tmp_x);


    // Iterate over hierarchy
    while (sm_n > 0) {
        // Iterate over blocks of 128 at this level
        for (uint32_t idx = warp_idx; idx < sm_n / 128; idx += num_warps) {
            // Move buffer
            Data *wx = x + idx * 128;
            // Local scan on the warp chunk
            Data end = warp_local_scan(wx);
            // If last thread write end back to tmp_x
            if (thread_idx == 31) {
                tmp_x[idx] = end;
            }
        }

        // Setup next level
        sm_n /= 128;
        std::swap(x, tmp_x);

        // Wait for this level to be done
        __syncthreads();
    }
}

template <typename Op>
__global__ void hierarchical_scan(size_t n, typename Op::Data *x, void *workspace) {
    using Data = typename Op::Data;
}

template <typename Op>
__global__ void local_scan_fix(size_t n, typename Op::Data *x, void *workspace) {
    using Data = typename Op::Data;
}

// Returns desired size of scratch buffer in bytes.
template <typename Op> size_t get_workspace_size(size_t n) {
    using Data = typename Op::Data;
    /* TODO: your CPU code here... */
    return 0;
}

// 'launch_scan'
//
// Input:
//
//   'n': Number of elements in the input array 'x'.
//
//   'x': Input array in GPU memory. The 'launch_scan' function is allowed to
//   overwrite the contents of this buffer.
//
//   'workspace': Scratch buffer in GPU memory. The size of the scratch buffer
//   in bytes is determined by 'get_workspace_size<Op>(n)'.
//
// Output:
//
//   Returns a pointer to GPU memory which will contain the results of the scan
//   after all launched kernels have completed. Must be either a pointer to the
//   'x' buffer or to an offset within the 'workspace' buffer.
//
//   The contents of the output array should be "partial reductions" of the
//   input; each element 'i' of the output array should be given by:
//
//     output[i] = Op::combine(x[0], x[1], ..., x[i])
//
//   where 'Op::combine(...)' of more than two arguments is defined in terms of
//   repeatedly combining pairs of arguments. Note that 'Op::combine' is
//   guaranteed to be associative, but not necessarily commutative, so
//
//        Op::combine(a, b, c)              // conceptual notation; not real C++
//     == Op::combine(a, Op::combine(b, c)) // real C++
//     == Op::combine(Op::combine(a, b), c) // real C++
//
//  but we don't necessarily have
//
//    Op::combine(a, b) == Op::combine(b, a) // not true in general!
//
template <typename Op>
typename Op::Data *launch_scan(
    size_t n,
    typename Op::Data *x, // pointer to GPU memory
    void *workspace       // pointer to GPU memory
) {
    using Data = typename Op::Data;
    
    // Kernel returns nothing
    gpu_scan<Op><<<1, 32>>>(n, x, workspace);

    // Approach: launch two kernels
    // First:
    // (1) Divide x across the 48 SMs
    // (2) Each SM computes the partial prefix across x/48
    // Intermediate:
    // (1) In the CPU code do the hierarchical scan of the 48 endpoints
    // Second:
    // (1) Seed each SM with the value from the hierarchical scan and recompute

    // Stages: (1) Local scan (GPU) (2) Hierarchical scan (CPU) (3) Local scan fix-up (GPU)

    return nullptr;
}

} // namespace scan_gpu

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

struct DebugRange {
    uint32_t lo;
    uint32_t hi;

    static constexpr uint32_t INVALID = 0xffffffff;

    static __host__ __device__ __forceinline__ DebugRange invalid() {
        return {INVALID, INVALID};
    }

    __host__ __device__ __forceinline__ bool operator==(const DebugRange &other) const {
        return lo == other.lo && hi == other.hi;
    }

    __host__ __device__ __forceinline__ bool operator!=(const DebugRange &other) const {
        return !(*this == other);
    }

    __host__ __device__ bool is_empty() const { return lo == hi; }

    __host__ __device__ bool is_valid() const { return lo != INVALID; }

    std::string to_string() const {
        if (lo == INVALID) {
            return "INVALID";
        } else {
            return std::to_string(lo) + ":" + std::to_string(hi);
        }
    }
};

struct DebugRangeConcatOp {
    using Data = DebugRange;

    static __host__ __device__ __forceinline__ Data identity() { return {0, 0}; }

    static __host__ __device__ __forceinline__ Data combine(Data a, Data b) {
        if (a.is_empty()) {
            return b;
        } else if (b.is_empty()) {
            return a;
        } else if (a.is_valid() && b.is_valid() && a.hi == b.lo) {
            return {a.lo, b.hi};
        } else {
            return Data::invalid();
        }
    }

    static std::string to_string(Data d) { return d.to_string(); }
};

struct SumOp {
    using Data = uint32_t;

    static __host__ __device__ __forceinline__ Data identity() { return 0; }

    static __host__ __device__ __forceinline__ Data combine(Data a, Data b) {
        return a + b;
    }

    static std::string to_string(Data d) { return std::to_string(d); }
};

constexpr size_t max_print_array_output = 1025;
static thread_local size_t total_print_array_output = 0;

template <typename Op> void print_array(size_t n, typename Op::Data const *x) {
    using Data = typename Op::Data;

    // copy 'x' from device to host if necessary
    hipPointerAttribute_t attr;
    CUDA_CHECK(hipPointerGetAttributes(&attr, x));
    auto x_host_buf = std::vector<Data>();
    Data const *x_host_ptr = nullptr;
    if (attr.type == hipMemoryTypeDevice) {
        x_host_buf.resize(n);
        x_host_ptr = x_host_buf.data();
        CUDA_CHECK(
            hipMemcpy(x_host_buf.data(), x, n * sizeof(Data), hipMemcpyDeviceToHost));
    } else {
        x_host_ptr = x;
    }

    if (total_print_array_output >= max_print_array_output) {
        return;
    }

    printf("[\n");
    for (size_t i = 0; i < n; i++) {
        auto s = Op::to_string(x_host_ptr[i]);
        printf("  [%zu] = %s,\n", i, s.c_str());
        total_print_array_output++;
        if (total_print_array_output > max_print_array_output) {
            printf("  ... (output truncated)\n");
            break;
        }
    }
    printf("]\n");

    if (total_print_array_output >= max_print_array_output) {
        printf("(Reached maximum limit on 'print_array' output; skipping further calls "
               "to 'print_array')\n");
    }

    total_print_array_output++;
}

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}

struct Results {
    double time_ms;
    double bandwidth_gb_per_sec;
};

enum class Mode {
    TEST,
    BENCHMARK,
};

template <typename Op>
Results run_config(Mode mode, std::vector<typename Op::Data> const &x) {
    // Allocate buffers
    using Data = typename Op::Data;
    size_t n = x.size();
    size_t workspace_size = scan_gpu::get_workspace_size<Op>(n);
    Data *x_gpu;
    Data *workspace_gpu;
    CUDA_CHECK(hipMalloc(&x_gpu, n * sizeof(Data)));
    CUDA_CHECK(hipMalloc(&workspace_gpu, workspace_size));
    CUDA_CHECK(hipMemcpy(x_gpu, x.data(), n * sizeof(Data), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));

    // Test correctness
    auto expected = std::vector<Data>(n);
    scan_cpu<Op>(n, x.data(), expected.data());
    auto out_gpu = scan_gpu::launch_scan<Op>(n, x_gpu, workspace_gpu);
    if (out_gpu == nullptr) {
        printf("'launch_scan' function not yet implemented (returned nullptr)\n");
        exit(1);
    }
    auto actual = std::vector<Data>(n);
    CUDA_CHECK(
        hipMemcpy(actual.data(), out_gpu, n * sizeof(Data), hipMemcpyDeviceToHost));
    for (size_t i = 0; i < n; ++i) {
        if (actual.at(i) != expected.at(i)) {
            auto actual_str = Op::to_string(actual.at(i));
            auto expected_str = Op::to_string(expected.at(i));
            printf(
                "Mismatch at position %zu: %s != %s\n",
                i,
                actual_str.c_str(),
                expected_str.c_str());
            if (n <= 128) {
                printf("Input:\n");
                print_array<Op>(n, x.data());
                printf("\nExpected:\n");
                print_array<Op>(n, expected.data());
                printf("\nActual:\n");
                print_array<Op>(n, actual.data());
            }
            exit(1);
        }
    }
    if (mode == Mode::TEST) {
        return {0.0, 0.0};
    }

    // Benchmark
    double target_time_ms = 200.0;
    double time_ms = benchmark_ms(
        target_time_ms,
        [&]() {
            CUDA_CHECK(
                hipMemcpy(x_gpu, x.data(), n * sizeof(Data), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
        },
        [&]() { scan_gpu::launch_scan<Op>(n, x_gpu, workspace_gpu); });
    double bytes_processed = n * sizeof(Data) * 2;
    double bandwidth_gb_per_sec = bytes_processed / time_ms / 1e6;

    // Cleanup
    CUDA_CHECK(hipFree(x_gpu));
    CUDA_CHECK(hipFree(workspace_gpu));

    return {time_ms, bandwidth_gb_per_sec};
}

std::vector<DebugRange> gen_debug_ranges(uint32_t n) {
    auto ranges = std::vector<DebugRange>();
    for (uint32_t i = 0; i < n; ++i) {
        ranges.push_back({i, i + 1});
    }
    return ranges;
}

template <typename Rng> std::vector<uint32_t> gen_random_data(Rng &rng, uint32_t n) {
    auto uniform = std::uniform_int_distribution<uint32_t>(0, 100);
    auto data = std::vector<uint32_t>();
    for (uint32_t i = 0; i < n; ++i) {
        data.push_back(uniform(rng));
    }
    return data;
}

template <typename Op, typename GenData>
void run_tests(std::vector<uint32_t> const &sizes, GenData &&gen_data) {
    for (auto size : sizes) {
        auto data = gen_data(size);
        printf("  Testing size %8u\n", size);
        run_config<Op>(Mode::TEST, data);
        printf("  OK\n\n");
    }
}

int main(int argc, char const *const *argv) {
    auto correctness_sizes = std::vector<uint32_t>{
        16,
        10,
        128,
        100,
        1024,
        1000,
        1 << 20,
        1'000'000,
        16 << 20,
        64 << 20,
    };

    auto rng = std::mt19937(0xCA7CAFE);

    printf("Correctness:\n\n");
    printf("Testing scan operation: debug range concatenation\n\n");
    run_tests<DebugRangeConcatOp>(correctness_sizes, gen_debug_ranges);
    printf("Testing scan operation: integer sum\n\n");
    run_tests<SumOp>(correctness_sizes, [&](uint32_t n) {
        return gen_random_data(rng, n);
    });

    printf("Performance:\n\n");

    size_t n = 64 << 20;
    auto data = gen_random_data(rng, n);

    printf("Benchmarking scan operation: integer sum, size %zu\n\n", n);

    // Warmup
    run_config<SumOp>(Mode::BENCHMARK, data);
    // Benchmark
    auto results = run_config<SumOp>(Mode::BENCHMARK, data);
    printf("  Time: %.2f ms\n", results.time_ms);
    printf("  Throughput: %.2f GB/s\n", results.bandwidth_gb_per_sec);

    return 0;
}